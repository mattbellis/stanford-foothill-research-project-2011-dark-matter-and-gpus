
#include <hip/hip_runtime.h>
#include<iostream>
#include<sstream>
#include<string>
#include<cstdlib>
#include<fstream>

__global__ void distance(float *x, float *y, float *z, int NUM_PART, float *dist)
{
 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;   
   int idx_dist = idx * NUM_PART; 

   float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
   float dist_x, dist_y, dist_z;

   for(int i=idx+1; i<NUM_PART; i++)
   {
      if(idx != i)
      {  
         dist_x = x_idx - x[i];
         dist_y = y_idx - y[i];
         dist_z = z_idx - z[i];
         dist[idx_dist + i] = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z); 
     }
   }
}

using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 3)
    {
        
        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    ifstream infile(argv[1]);

    FILE *output_dist;
    
    output_dist = fopen(argv[2], "w");
    
    if(!output_dist)
    {
         perror("Error opening file");
         return 1;
    }


   // ofstream outile_distances(argv[2])

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    
//    char axis_titles[256];
    char dummy;

  
  if(infile.good())
    {
        infile >> NUM_PARTICLES;
    
//        for(int m=0; m <3; m++)
   //        infile >> axis_titles >> axis_titles;

    }
    else
     {
        printf("Couldn't open the file for input.\n");
        exit(1);
    }



  //  fscanf(infile, "%d", &NUM_PARTICLES);
  //  fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);
    
    int size = NUM_PARTICLES * sizeof(float);    
        
    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float *)malloc(size * size);
/*   
    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
    }
*/
    for(int i=0; i<NUM_PARTICLES; i++)
    {

       infile >> pos_x[i] >> dummy;
       infile >> pos_y[i] >> dummy;
       infile >> pos_z[i] >> dummy;
    }
    ////////////////////////////////////////////////////////////////////////////
   
    dim3 grid,block;
    block.x = 1000;

    grid.x = NUM_PARTICLES/block.x;
 
   
    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
    hipMalloc((void **) &dev_dist, size * size);

    if (0==h_dist || 0==dev_dist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset(dev_pos_x, 0, size);
    hipMemset(dev_pos_y, 0, size);
    hipMemset(dev_pos_z, 0, size);

    hipMemset(dev_dist, 0, size*size);
    
    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );
   
    
    distance<<<block, grid>>>(dev_pos_x, dev_pos_y, dev_pos_z, NUM_PARTICLES, dev_dist);
 
    hipMemcpy(h_dist, dev_dist, size * size, hipMemcpyDeviceToHost );

    fprintf(output_dist, "%s %i \n", "GPU ", NUM_PARTICLES);
 
    for(int k=0; k< NUM_PARTICLES * NUM_PARTICLES; k++)
    {
       if(h_dist[k] > 0)
         fprintf(output_dist, "%e \n", h_dist[k]);
    }
    
    fclose(output_dist);

    free(pos_x);
    free(pos_y);
    free(pos_z);
    
    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

   return 0;
}  
