
#include <hip/hip_runtime.h>
#include<iostream>
#include<sstream>
#include<string>
#include<cstdlib>
#include<fstream>

__global__ void distance(float *x, float *y, float *z, int NUM_PART, float *dist)
{
   float posx, posy, posz;
   
   
   for(int i=0; i<NUM_PART-1; i++)
   {
      posx = x[i+1] - x[i];
      posy = y[i+1] - y[i];
      posz = z[i+1] - z[i];
      dist[i] = sqrt(posx * posx + posy * posy + posz * posz); 
   }
       
}


using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 2)
    {
        
        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    ifstream infile(argv[1]);

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    string axis_titles;
  
    
    string dummy;

    if(infile.good())
    {
        infile >> NUM_PARTICLES;
    
        for(int m=0; m <3; m++)
           infile >> axis_titles >> axis_titles;


    }
    else
     {
        printf("Couldn't open the file for input.\n");
        exit(1);
    }

 
    int size = NUM_PARTICLES * sizeof(float);    
        
    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float *)malloc(size);
   
    for(int i=0; i<NUM_PARTICLES; i++)
    {

       infile >> pos_x[i] >> dummy;
       infile >> pos_y[i] >> dummy;
       infile >> pos_z[i] >> dummy;
    }

    ////////////////////////////////////////////////////////////////////////////
    

    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
    hipMalloc((void **) &dev_dist, size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );
    
    printf("%i\n", NUM_PARTICLES);
    for(int k=0; k< NUM_PARTICLES; k++)
       printf("%e ", pos_x[k]);    

    distance<<<NUM_PARTICLES, 1 >>>(dev_pos_x, dev_pos_y, dev_pos_z, NUM_PARTICLES, dev_dist);

    
    hipMemcpy(h_dist, dev_dist, size, hipMemcpyDeviceToHost );
    
    free(pos_x);
    free(pos_y);
    free(pos_z);
    
    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

   return 0;
}  