
#include <hip/hip_runtime.h>
#include<iostream>
#include<sstream>
#include<string>
#include<cstdlib>
#include<fstream>

__global__ void copy_coordinates(float *dev_x, float *dev_y, float *dev_z)
{
      
}


using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z;
    float *pos_x, *pos_y, *pos_z;

    int NUM_PARTICLES;

    if (argc < 3)
    {
        cerr << endl;
        cerr << "Must pass in cluster_data file  on command line!" << endl;
        cerr << "Usage: " << argv[0] << " <cluster_data file> <distances file> " << endl;
        cerr << endl;
        exit(1);
    }

    ifstream infile(argv[1]);

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    string axis_titles;
  
    
    string dummy;

    if(infile.good())
    {
        infile >> NUM_PARTICLES;
    
        for(int m=0; m <3; m++)
           infile >> axis_titles >> axis_titles;


    }
    else
     {
        cerr << "Couldn't open the file for input." << endl;
        exit(1);
    }

 
    int size = NUM_PARTICLES * sizeof(float);    
        
    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
   
    for(int i=0; i<NUM_PARTICLES; i++)
    {

       infile >> pos_x[i] >> dummy;
       infile >> pos_y[i] >> dummy;
       infile >> pos_z[i] >> dummy;
    }

    ////////////////////////////////////////////////////////////////////////////
    

    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );
    
for(int k=0; k< NUM_PARTICLES; k++)
   cerr << pos_x[k];    


free(pos_x);
    free(pos_y);
    free(pos_z);
    
    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

   return 0;
}  
