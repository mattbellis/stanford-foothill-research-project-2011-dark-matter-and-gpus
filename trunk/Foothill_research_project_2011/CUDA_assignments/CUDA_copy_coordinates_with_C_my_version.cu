
#include <hip/hip_runtime.h>
#include<iostream>
#include<sstream>
#include<string>
#include<cstdlib>
#include<fstream>

__global__ void distance(float *x, float *y, float *z, int NUM_PART, float *dist)
{

   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int idx_dist = idx * NUM_PART;

   float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
   float dist_x, dist_y, dist_z;

   for(int i=idx+1; i<NUM_PART; i++)
   {
      if(idx != i)
      {
         dist_x = x_idx - x[i];
         dist_y = y_idx - y[i];
         dist_z = z_idx - z[i];
         dist[idx_dist + i] = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
     }
   }
}

using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    //ifstream infile(argv[1]);
    FILE *infile;
    infile = fopen(argv[1],"r");

    FILE *output_dist;

    output_dist = fopen(argv[2], "w");

    if(!output_dist)
    {
         perror("Error opening file");
         return 1;
    }

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

   // char axis_titles[256];
    char dummy[256];

    fscanf(infile, "%d", &NUM_PARTICLES);
  //  fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float*)malloc(size * size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
    }

    ////////////////////////////////////////////////////////////////////////////

    dim3 grid,block;
    block.x = 512;
    grid.x = NUM_PARTICLES/block.x;


    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
    hipMalloc((void **) &dev_dist, size * size);

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);
    hipMemset(dev_dist,0,size*size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );


    distance<<<block, grid >>>(dev_pos_x, dev_pos_y, dev_pos_z, NUM_PARTICLES, dev_dist);

    //cudaMemset(dev_dist,1.0,size*size);
    hipMemcpy(h_dist, dev_dist, size * size, hipMemcpyDeviceToHost );
    
    fprintf(output_dist, "%s %i \n", "GPU ", NUM_PARTICLES);

    for(int k=0; k< NUM_PARTICLES * NUM_PARTICLES; k++)
    {
       if(h_dist[k] > 0)
         fprintf(output_dist, "%e \n", h_dist[k]);
    }

    fclose(output_dist);   

    free(pos_x);
    free(pos_y);
    free(pos_z);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

    return 0;
}  
