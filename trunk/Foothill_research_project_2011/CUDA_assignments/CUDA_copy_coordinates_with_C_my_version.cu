#include "hip/hip_runtime.h"
#include<stdio.h>
//nclude<sstream>
#include<string.h>
#include<stdlib.h>
//nclude<fstream>

__global__ void distance(float *x, float *y, float *z, int xind, int yind)// float *dist)
{


   int idx = xind * blockDim.x + yind;
 //  int idx = blockIdx.x * blockDim.x + threadIdx.x;
//   int idx_dist = idx * NUM_PART;
   float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
   float dist_x, dist_y, dist_z, dist;

   for(int i=idx+1; i<blockDim.x; i++)
   {
      if(idx != i)
      {
         dist_x = x_idx - x[i];
         dist_y = y_idx - y[i];
         dist_z = z_idx - z[i];
         dist = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
//         dist[idx_dist + i] = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
     }
   }
}

using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    //ifstream infile(argv[1]);
    FILE *infile;
    infile = fopen(argv[1],"r");

   // FILE *output_dist;

   // output_dist = fopen(argv[2], "w");

   // if(!output_dist)
   // {
   //      perror("Error opening file");
   //      return 1;
   // }

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

   // char axis_titles[256];
    char dummy[256];

    fscanf(infile, "%d", &NUM_PARTICLES);
  //  fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float*)malloc(size * size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
    }

    ////////////////////////////////////////////////////////////////////////////

    dim3 grid, block;
    block.x = 512;
    grid.x = 127; //NUM_PARTICLES/block.x;


    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
  //  hipMalloc((void **) &dev_dist, size * size);

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x)// || 0==dev_dist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);
   // hipMemset(dev_dist,0,size*size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );

   int x, y;
   int block_num = NUM_PARTICLES / block.x;

   for(int k = 0; k < block_num; k++)
   {
      y = k* block.x;
      for(int j = 0; j < block_num; j++)
      {
         { 
            x = j * block.x; 
            distance<<<grid, block >>>(dev_pos_x, dev_pos_y, dev_pos_z, x, y);//, dev_dist);
         }
      }
    //hipMemset(dev_dist,1.0,size*size);
   // hipMemcpy(h_dist, dev_dist, size * size, hipMemcpyDeviceToHost );
    
  //  fprintf(output_dist, "%s %i \n", "GPU ", NUM_PARTICLES);

  //  for(int k=0; k< NUM_PARTICLES * NUM_PARTICLES; k++)
  //  {
  //     if(h_dist[k] > 0)
  //       fprintf(output_dist, "%e \n", h_dist[k]);
  //  }

  //  fclose(output_dist);   

    free(pos_x);
    free(pos_y);
    free(pos_z);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

    return 0;
}  
