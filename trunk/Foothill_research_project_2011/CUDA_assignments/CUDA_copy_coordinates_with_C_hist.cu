
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<stdlib.h>

using namespace std;

#define SUBMATRIX_SIZE 1000
#define NUM_BIN 100
#define HIST_MIN 0.0
#define HIST_MAX 3e9

////////////////////////////////////////////////////////////////////////
__global__ void distance(float *x, float *y, float *z, int xind, int yind, int *dev_hist)
{


    //int idx = xind * blockDim.x + yind;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_idx = idx;
    idx += xind;

    float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
    float dist_x, dist_y, dist_z, dist;

    //int max = SUBMATRIX_SIZE*

    int ymax = yind + SUBMATRIX_SIZE;
    int bin_index; 
    int offset = 0;

    for(int i=yind; i<ymax; i++)
    {
        //if(idx != i)
        if(idx > i)
        {
            dist_x = x_idx - x[i];
            dist_y = y_idx - y[i];
            dist_z = z_idx - z[i];
            dist = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
            if(dist < HIST_MIN)
                bin_index = 0; 
            else if(dist >= HIST_MAX)
                bin_index = NUM_BIN + 1;
            else
                bin_index = int(((dist - HIST_MIN) * NUM_BIN / HIST_MAX) +1);    
                //bin_index = 5;

            //dev_hist[bin_index]++;

            offset = ((NUM_BIN+2)*thread_idx);
            bin_index += offset;

           //dev_hist[i] = xind;
           //dev_hist[i+idx] = idx;
           dev_hist[bin_index]++;

           //dev_hist[0+offset] = blockDim.x;
           //dev_hist[1+offset] = blockIdx.x;
           //dev_hist[2+offset] = threadIdx.x;
           //dev_hist[3+offset] = thread_idx;
           //dev_hist[4+offset] = idx;
           //dev_hist[5+offset] = yind;
           //dev_hist[6+offset] = ymax;
        }
    }
    //dev_hist[0 + (threadIdx.x*12)] = threadIdx.x;
    /*
    for (int i=0;i<10;i++)
    {
        offset = i*12;
        //offset = 0.0;
        dev_hist[threadIdx.x+offset] = threadIdx.x;
        //dev_hist[offset] = threadIdx.x;
        //dev_hist[offset] = 999;
    }
    */
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z;
    float *pos_x, *pos_y, *pos_z;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    FILE *infile;
    infile = fopen(argv[1],"r");


    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    // char axis_titles[256];
    char dummy[256];

    fscanf(infile, "%d", &NUM_PARTICLES);
    //fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    
    printf("# particles: %d\n",NUM_PARTICLES);

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
        //printf("%e %s %e %s %e %s\n", pos_x[i], dummy, pos_y[i], dummy, pos_z[i], dummy);
    }
    ////////////////////////////////////////////////////////////////////////////
    //allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;
    int size_hist = SUBMATRIX_SIZE * (NUM_BIN+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("size_hist: %d\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    int *hist_array;

    hist_array =  (int*)malloc((NUM_BIN+2) * sizeof(int));
    memset(hist_array, 0, (NUM_BIN+2)*sizeof(int)); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    grid.x = 10;
    block.x = SUBMATRIX_SIZE/grid.x; //NUM_PARTICLES/block.x;
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices = NUM_PARTICLES / SUBMATRIX_SIZE;


    int bin_index = 0;
    for(int k = 0; k < num_submatrices; k++)
    {
        y = k*SUBMATRIX_SIZE;
        printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices; j++)
        {
                x = j *SUBMATRIX_SIZE; 

                //printf("----\n");
                //printf("%d %d\t\t%d %d\n",k,y,j,x);
                //printf("----\n");

                hipMemset(dev_hist,0,size_hist_bytes);

                distance<<<grid,block>>>(dev_pos_x, dev_pos_y, dev_pos_z, x, y, dev_hist);
                hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);


                for(int m=0; m<size_hist; m++)
                {

                    bin_index = m%(NUM_BIN+2);
                    //if(bin_index == 0)
                        //printf("\n");

                    //printf("%3i:%3i ", m, hist[m]);
                    //printf("%3i ", hist[m]);

                    hist_array[bin_index] += hist[m];
                }    
                //printf("\n");

        }
    }

    // cudaMemcpy(hist, dev_hist, size_hist, cudaMemcpyDeviceToHost);
    /*
    for(int j=0; j<NUM_BIN+2; j++)
        for(int i=0; i<SUBMATRIX_SIZE; i++)
            hist_array[j] += hist[i*(NUM_BIN + 2)+j];
    */

    unsigned long total = 0;
    for(int k=0; k<NUM_BIN+2; k++)
    {
        printf("%i \n", hist_array[k]);
        total += hist_array[k];
    }
    printf("total: %lu \n", total);

    free(pos_x);
    free(pos_y);
    free(pos_z);
    free(hist);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////
