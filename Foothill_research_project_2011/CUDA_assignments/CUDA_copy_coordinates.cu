
#include <hip/hip_runtime.h>
#include<iostream>
#include<sstream>
#include<string>
#include<cstdlib>
#include<fstream>

__global__ void distance(float *x, float *y, float *z, int NUM_PART, float *dist)
{
   float posx, posy, posz;
 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;   
   int idx_dist = idx * (NUM_PART-1); 
   for(int i=0; i<NUM_PART; i++)
   {
      if(idx != i)
      {  
         posx = x[idx] - x[i];
         posy = y[idx] - y[i];
         posz = z[idx] - z[i];
         dist[idx_dist + i] = sqrt(posx * posx + posy * posy + posz * posz); 
      }
   }
}


using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 2)
    {
        
        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    ifstream infile(argv[1]);

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    string axis_titles;
  
    
    string dummy;

    if(infile.good())
    {
        infile >> NUM_PARTICLES;
    
        for(int m=0; m <3; m++)
           infile >> axis_titles >> axis_titles;

    }
    else
     {
        printf("Couldn't open the file for input.\n");
        exit(1);
    }

 
    int size = NUM_PARTICLES * sizeof(float);    
        
    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float *)malloc(size * size);
   
    for(int i=0; i<NUM_PARTICLES; i++)
    {

       infile >> pos_x[i] >> dummy;
       infile >> pos_y[i] >> dummy;
       infile >> pos_z[i] >> dummy;
    }

    ////////////////////////////////////////////////////////////////////////////
   
    dim3 grid,block;
    block.x = 1;
    grid.x = NUM_PARTICLES/block.x;
 
   
    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
    hipMalloc((void **) &dev_dist, size * size);

    if (0==h_dist || 0==dev_dist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );
    
    printf("%i\n", NUM_PARTICLES);
    for(int k=0; k< NUM_PARTICLES; k++)
       printf("%e ", pos_x[k]);    

    distance<<<block, NUM_PARTICLES >>>(dev_pos_x, dev_pos_y, dev_pos_z, NUM_PARTICLES, dev_dist);
 
    hipMemcpy(h_dist, dev_dist, size * size, hipMemcpyDeviceToHost );
 
    printf("%s\n", "distances");
    for(int k=0; k< NUM_PARTICLES * NUM_PARTICLES; k++)
       printf("%e ", h_dist[k]);
    
    free(pos_x);
    free(pos_y);
    free(pos_z);
    
    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

   return 0;
}  
