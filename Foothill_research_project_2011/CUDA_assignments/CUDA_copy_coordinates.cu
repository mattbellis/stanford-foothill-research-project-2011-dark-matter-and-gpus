#include "hip/hip_runtime.h"
#include<iostream>
#include<sstream>
#include<string>
#include<cstdlib>
#include<fstream>

__global__ void copy_coordinates(float *dev_x, float *dev_y, float *dev_z)
{
  
}


using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z;

    int NUM_PARTICLES;

    if (argc < 3)
    {
        cerr << endl;
        cerr << "Must pass in cluster_data file  on command line!" << endl;
        cerr << "Usage: " << argv[0] << " <cluster_data file> <distances file> " << endl;
        cerr << endl;
        exit(1);
    }

    ifstream infile(argv[1]);

    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////
    infile >> NUM_PARTICLES;

    string axis_titles;
    for(int m=0; m <3; m++)
       infile >> axis_titles >> axis_titles;

    float pos[NUM_PARTICLES][3];

    string dummy;

    if(infile.good())
    {
        for(int i=0; i<NUM_PARTICLES; i++)
        {

            infile >> pos[i][0] >> dummy;
            infile >> pos[i][1] >> dummy;
            infile >> pos[i][2] >> dummy;
        }

    }
    else
     {
        cerr << "Couldn't open the file for input." << endl;
        exit(1);
    }
    ////////////////////////////////////////////////////////////////////////////
    
    int size = NUM_PARTICLES * sizeof(float);    

    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );

    hipMemcpy(dev_pos_x, pos[NUM_PARTICLES][0], size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos[NUM_PARTICLES][1], size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos[NUM_PARTICLES][2], size, hipMemcpyHostToDevice );
    
    
    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

   return 0;
}  
