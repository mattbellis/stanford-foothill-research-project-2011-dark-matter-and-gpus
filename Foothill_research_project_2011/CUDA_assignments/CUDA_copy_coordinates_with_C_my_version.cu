
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<stdlib.h>

using namespace std;

#define SUBMATRIX_SIZE 3000

////////////////////////////////////////////////////////////////////////
__global__ void distance(float *x, float *y, float *z, int xind, int yind)// float *dist)
{


   //int idx = xind * blockDim.x + yind;
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   idx += xind;

   float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
   float dist_x, dist_y, dist_z, dist;


   //int max = SUBMATRIX_SIZE*

    int ymax = yind + SUBMATRIX_SIZE;
   for(int i=yind; i<ymax; i++)
   {
      if(idx != i)
      {
         dist_x = x_idx - x[i];
         dist_y = y_idx - y[i];
         dist_z = z_idx - z[i];
         dist = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
//         dist[idx_dist + i] = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
     }
   }
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    //ifstream infile(argv[1]);
    FILE *infile;
    infile = fopen(argv[1],"r");


    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

   // char axis_titles[256];
    char dummy[256];

    fscanf(infile, "%d", &NUM_PARTICLES);
  //  fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    
    printf("# particles: %d\n",NUM_PARTICLES);

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float*)malloc(size * size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
        //printf("%e %s %e %s %e %s\n", pos_x[i], dummy, pos_y[i], dummy, pos_z[i], dummy);
    }
    

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    block.x = 100;
    grid.x = SUBMATRIX_SIZE/block.x; //NUM_PARTICLES/block.x;
    ////////////////////////////////////////////////////////////////////////////


    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
  //  cudaMalloc((void **) &dev_dist, size * size);

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x)// || 0==dev_dist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);
   // cudaMemset(dev_dist,0,size*size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_dist, h_dist, size * size, hipMemcpyHostToDevice );

   int x, y;
   int num_submatrices = NUM_PARTICLES / SUBMATRIX_SIZE;

   for(int k = 0; k < num_submatrices; k++)
   {
      y = k*SUBMATRIX_SIZE;
      for(int j = 0; j < num_submatrices; j++)
      {
         { 
            x = j *SUBMATRIX_SIZE; 
        //    printf("x: %d\ty: %d\n",x,y);
            distance<<<grid, block >>>(dev_pos_x, dev_pos_y, dev_pos_z, x, y);//, dev_dist);
         }
      }
   }
    //cudaMemset(dev_dist,1.0,size*size);
   // cudaMemcpy(h_dist, dev_dist, size * size, cudaMemcpyDeviceToHost );
    
  //  fprintf(output_dist, "%s %i \n", "GPU ", NUM_PARTICLES);

  //  for(int k=0; k< NUM_PARTICLES * NUM_PARTICLES; k++)
  //  {
  //     if(h_dist[k] > 0)
  //       fprintf(output_dist, "%e \n", h_dist[k]);
  //  }

  //  fclose(output_dist);   

    free(pos_x);
    free(pos_y);
    free(pos_z);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

    return 0;
}  
////////////////////////////////////////////////////////////////////////
