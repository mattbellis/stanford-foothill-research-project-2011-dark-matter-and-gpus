
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<stdlib.h>

using namespace std;

#define SUBMATRIX_SIZE 10
#define NUM_BIN 10
#define MIN 0.0
#define MAX 35.0  

////////////////////////////////////////////////////////////////////////
__global__ void distance(float *x, float *y, float *z, int xind, int yind, int *dev_hist)
{


    //int idx = xind * blockDim.x + yind;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_idx = idx;
    idx += xind;

    float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
    float dist_x, dist_y, dist_z, dist;

    //int max = SUBMATRIX_SIZE*

    int ymax = yind + SUBMATRIX_SIZE;
    int bin_index,  bin = idx * (NUM_BIN + 2); 
    int offset = 0;

    /*
    for(int i=yind; i<ymax; i++)
    {
        if(idx != i)
        {
            dist_x = x_idx - x[i];
            dist_y = y_idx - y[i];
            dist_z = z_idx - z[i];
            dist = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
            if(dist < MIN)
                bin_index = 0; 
            else if(dist >= MAX)
                bin_index = NUM_BIN + 1;
            else
                //bin_index = int(((dist - MIN) * NUM_BIN / MAX) +1);    
                bin_index = 5;

            //dev_hist[bin_index]++;

            offset = ((NUM_BIN+2)*thread_idx);
            bin_index += offset;

           //dev_hist[i] = xind;
           //dev_hist[i+idx] = idx;
           //dev_hist[bin_index]++;
           dev_hist[0+offset] = blockDim.x;
           dev_hist[1+offset] = blockIdx.x;
           dev_hist[2+offset] = threadIdx.x;
           dev_hist[3+offset] = thread_idx;
           dev_hist[4+offset] = idx;
           dev_hist[5+offset] = yind;
           dev_hist[6+offset] = ymax;
        }
    }
    */
    //dev_hist[0 + (threadIdx.x*12)] = threadIdx.x;
    for (int i=0;i<10;i++)
    {
        offset = threadIdx.x*12;
        dev_hist[threadIdx.x+offset] = threadIdx.x;
    }
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z;
    float *pos_x, *pos_y, *pos_z;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    FILE *infile;
    infile = fopen(argv[1],"r");


    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    // char axis_titles[256];
    char dummy[256];

    fscanf(infile, "%d", &NUM_PARTICLES);
    //fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    
    printf("# particles: %d\n",NUM_PARTICLES);

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
        //printf("%e %s %e %s %e %s\n", pos_x[i], dummy, pos_y[i], dummy, pos_z[i], dummy);
    }
    ////////////////////////////////////////////////////////////////////////////
    //allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;
    int size_hist = SUBMATRIX_SIZE * (NUM_BIN+2);

    hist = (int*)malloc(size_hist * sizeof(int));
    memset(hist, 0, size_hist);

    hipMalloc((void **) &dev_hist, size_hist);
    hipMemset(dev_hist, 0, size_hist);

    int *hist_array;

    hist_array =  (int*)malloc((NUM_BIN+2) * sizeof(int));
    memset(hist_array, 0, (NUM_BIN+2)); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    grid.x = 1;
    block.x = SUBMATRIX_SIZE/grid.x; //NUM_PARTICLES/block.x;
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices = NUM_PARTICLES / SUBMATRIX_SIZE;


    for(int k = 0; k < num_submatrices; k++)
    {
        y = k*SUBMATRIX_SIZE;
        for(int j = 0; j < num_submatrices; j++)
        {
                x = j *SUBMATRIX_SIZE; 

                printf("----\n");
                printf("%d %d\t\t%d %d\n",k,y,j,x);
                printf("----\n");

                hipMemset(dev_hist,0,size_hist);

                distance<<<grid,block>>>(dev_pos_x, dev_pos_y, dev_pos_z, x, y, dev_hist);
                hipMemcpy(hist, dev_hist, size_hist, hipMemcpyDeviceToHost);


                for(int m=0; m<size_hist; m++)
                {
                    if((m%12) == 0)
                        printf("\n");

                    //printf("%3i:%3i ", m, hist[m]);
                    printf("%3i ", hist[m]);
                }    
                printf("\n");
        }
    }

    // cudaMemcpy(hist, dev_hist, size_hist, cudaMemcpyDeviceToHost);
    for(int j=0; j<NUM_BIN+2; j++)
        for(int i=0; i<SUBMATRIX_SIZE; i++)
            hist_array[j] += hist[i*(NUM_BIN + 2)+j];

    int total = 0;
    for(int k=0; k<NUM_BIN+2; k++)
    {
        printf("%i \n", hist_array[k]);
        total += hist_array[k];
    }
    printf("total: %i \n", total);

    free(pos_x);
    free(pos_y);
    free(pos_z);
    free(hist);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////
