
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<stdlib.h>

using namespace std;

#define SUBMATRIX_SIZE 10
#define NUM_BIN 10
#define MIN 0.0
#define MAX 100.0  

////////////////////////////////////////////////////////////////////////
__global__ void distance(float *x, float *y, float *z, int xind, int yind, int *dev_hist)
{


   //int idx = xind * blockDim.x + yind;
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   idx += xind;

   float x_idx = x[idx], y_idx =y[idx], z_idx = z[idx];
   float dist_x, dist_y, dist_z, dist;
   
   //int max = SUBMATRIX_SIZE*

   int ymax = yind + SUBMATRIX_SIZE;
   int bin_index,  bin = idx * (NUM_BIN + 2); 

   for(int i=yind; i<ymax; i++)
   {
      if(idx != i)
      {
         dist_x = x_idx - x[i];
         dist_y = y_idx - y[i];
         dist_z = z_idx - z[i];
         dist = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);

         if(dist < MIN)
            bin_index = bin; 
         else if(dist >= MAX)
            bin_index = bin + NUM_BIN + 1;
         else
            bin_index = bin + int(((dist - MIN) * NUM_BIN / MAX) +1);    
   
         dev_hist[bin_index]++;

//for(int i=0; i<100; i++)
//dev_hist[i] =1;
     }
   }
}

////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z;
    float *pos_x, *pos_y, *pos_z;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    FILE *infile;
    infile = fopen(argv[1],"r");


    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    // char axis_titles[256];
    char dummy[256];

    fscanf(infile, "%d", &NUM_PARTICLES);
    //fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    
    printf("# particles: %d\n",NUM_PARTICLES);

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
        //printf("%e %s %e %s %e %s\n", pos_x[i], dummy, pos_y[i], dummy, pos_z[i], dummy);
    }
    ////////////////////////////////////////////////////////////////////////////
    //allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;
    int size_hist = SUBMATRIX_SIZE * (NUM_BIN+2);

    hist = (int*)malloc(size_hist * sizeof(int));
    memset(hist, 0, size_hist);

    hipMalloc((void **) &dev_hist, size_hist);
    hipMemset(dev_hist, 0, size_hist);
  
    int *hist_array;
   
    hist_array =  (int*)malloc((NUM_BIN+2) * sizeof(int));
    memset(hist_array, 0, (NUM_BIN+2)); 
    
    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    block.x = 10;
    grid.x = SUBMATRIX_SIZE/block.x; //NUM_PARTICLES/block.x;
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices = NUM_PARTICLES / SUBMATRIX_SIZE;


 /*   for(int k = 0; k < num_submatrices; k++)
    {
       y = k*SUBMATRIX_SIZE;
       for(int j = 0; j < num_submatrices; j++)
       {
          { 
             x = j *SUBMATRIX_SIZE; 
*/
             distance<<<grid, block >>>(dev_pos_x, dev_pos_y, dev_pos_z, 0, 0, dev_hist);
               hipMemcpy(hist, dev_hist, size_hist, hipMemcpyDeviceToHost);

for(int m=0; m<size_hist; m++)
{
   if((m%12) == 0)
     printf("\n");

   printf("%i ", hist[m]);
}    
printf("\n");

/*
          }
       }
    }
*/
   // cudaMemcpy(hist, dev_hist, size_hist, cudaMemcpyDeviceToHost);
    for(int j=0; j<NUM_BIN+2; j++)
      for(int i=0; i<SUBMATRIX_SIZE; i++)
          hist_array[j] += hist[i*(NUM_BIN + 2)+j];

    for(int k=0; k<NUM_BIN+2; k++)
       printf("%i \n", hist_array[k]);

    free(pos_x);
    free(pos_y);
    free(pos_z);
    free(hist);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////
