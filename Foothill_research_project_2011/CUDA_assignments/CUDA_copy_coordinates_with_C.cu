
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>




__global__ void distance(float *x, float *y, float *z, int NUM_PART, float *dist)
{
    float posx, posy, posz;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;   
    int idx_dist = idx * (NUM_PART); 
    for(int i=0; i<NUM_PART; i++)
    {
        if(idx != i)
        {  
            posx = x[idx] - x[i];
            posy = y[idx] - y[i];
            posz = z[idx] - z[i];
            dist[idx_dist + i] = sqrt(posx * posx + posy * posy + posz * posz); 
        }
    }
}


using namespace std;

int main(int argc, char **argv)
{

    float *dev_pos_x, *dev_pos_y, *dev_pos_z, *dev_dist;
    float *pos_x, *pos_y, *pos_z, *h_dist;

    int NUM_PARTICLES;

    if (argc < 2)
    {

        printf("\nMust pass in cluster_data file  on command line!\n");
        printf("\nUsage: ", argv[0] );
        printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    //ifstream infile(argv[1]);
    FILE *infile;
    infile = fopen(argv[1],"r");


    //////////////////////////////////////////////////////////////////////
    // Read in the cluster_data file
    ////////////////////////////////////////////////////////////////////////////

    char axis_titles[256];
    char dummy[256];

    /*
       if(infile.good())
       {
       infile >> NUM_PARTICLES;

       for(int m=0; m <3; m++)
       infile >> axis_titles >> axis_titles;

       }
       else
       {
       printf("Couldn't open the file for input.\n");
       exit(1);
       }
     */

    fscanf(infile, "%d", &NUM_PARTICLES);
    fscanf(infile, "%s %s %s %s", &axis_titles, &dummy, &axis_titles, &dummy);

    int size = NUM_PARTICLES * sizeof(float);    

    pos_x = (float*)malloc(size);
    pos_y = (float*)malloc(size);
    pos_z = (float*)malloc(size);
    h_dist = (float *)malloc(size * size);

    for(int i=0; i<NUM_PARTICLES; i++)
    {
        fscanf(infile, "%e %s %e %s %e %s", &pos_x[i], &dummy, &pos_y[i], &dummy, &pos_z[i], &dummy);
    }

    ////////////////////////////////////////////////////////////////////////////

    dim3 grid,block;
    block.x = 1;
    grid.x = NUM_PARTICLES/block.x;


    hipMalloc((void **) &dev_pos_x, size );
    hipMalloc((void **) &dev_pos_y, size );
    hipMalloc((void **) &dev_pos_z, size );
    hipMalloc((void **) &dev_dist, size * size);

    // Check to see if we allocated enough memory.
    if (0==dev_pos_z || 0==dev_pos_y|| 0==dev_pos_x)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }


    // Initialize array to all 0's
    hipMemset(dev_pos_x,0,size);
    hipMemset(dev_pos_z,0,size);
    hipMemset(dev_pos_y,0,size);
    hipMemset(dev_dist,0,size*size);

    hipMemcpy(dev_pos_x, pos_x, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_y, pos_y, size, hipMemcpyHostToDevice );
    hipMemcpy(dev_pos_z, pos_z, size, hipMemcpyHostToDevice );

    printf("%i\n", NUM_PARTICLES);
    for(int k=0; k< NUM_PARTICLES; k++)
        printf("%e ", pos_x[k]);    

    distance<<<block, NUM_PARTICLES >>>(dev_pos_x, dev_pos_y, dev_pos_z, NUM_PARTICLES, dev_dist);

    //cudaMemset(dev_dist,1.0,size*size);
    hipMemcpy(h_dist, dev_dist, size * size, hipMemcpyDeviceToHost );

    printf("\n%s\n", "distances");
    for(int k=0; k< NUM_PARTICLES * NUM_PARTICLES; k++)
    {
        printf("%e\n", h_dist[k]);
    }

    free(pos_x);
    free(pos_y);
    free(pos_z);

    hipFree(dev_pos_x);
    hipFree(dev_pos_y);  
    hipFree(dev_pos_z);

    return 0;
}  
